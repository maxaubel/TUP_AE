#include <hip/hip_runtime.h>
#include <bits/stdc++.h>
#include <algorithm>
#include <iostream>
#include <regex>
#include <vector>
#include <cmath>
#include <chrono>

using namespace std;
using namespace std::chrono;

vector<vector <int>> dist;
vector<vector <int>> opponents;
vector<vector <int>> home_venues;

int q1;
int q2;

int POPULATION_SIZE;

# define HOME_VENUE_PENALIZATION 5000
# define Q1_PENALIZATION 5000
# define Q2_PENALIZATION 5000

// clean a string from file
string str_cleaner(string line) {
    line.erase(remove(line.begin(), line.end(), '['), line.end());
    line.erase(remove(line.begin(), line.end(), ']'), line.end());
    return line;
}

// tokenize a line from the text file
int* tokenize(string line, int nTeams) {

    line = str_cleaner(line);
    string* parsed_line = new string[nTeams];
    int* dist_values = new int[nTeams];
    stringstream ssin(line);
    int i = 0;

    while (ssin.good() && i < nTeams) {
        ssin >> parsed_line[i];
        ++i;
    }

    for (int i = 0; i < nTeams; ++i)
        dist_values[i] = stoi(parsed_line[i]);

    return dist_values;
}

void printf_vector(vector<int> v) {
    int i=0;
    for (; i < v.size()-1; i++)
        printf("%d, ", v[i]);
    printf("%d\n", v[i]);
    return;
}

